#include "hip/hip_runtime.h"
#include "kernel.hu"
const int TILE_SIZE  = 32;
const int BLOCK_SIZE = 1024;
__global__ void ACGPUSimple(const int* tr, const char* text, int* occur, const int M, const int L, const int charSetSize) {
  int idx         = blockIdx.x * blockDim.x;
  int threadStart = idx + threadIdx.x * TILE_SIZE;
  int threadEnd   = threadStart + TILE_SIZE + M - 1;

  int state = 0;
  for (int i = threadStart; i < threadEnd; i++) {
    state = tr[state * charSetSize + text[i]];
    atomicAdd(&occur[state], 1); // Optimizable
  }
}
void ACGPUSimpleLaunch(const int* tr, const char* text, int* occur, const int M, const int L, const int charSetSize) {
  int blockNum = (L - 1) / (BLOCK_SIZE * TILE_SIZE) + 1;
  ACGPUSimple<<<blockNum, BLOCK_SIZE>>>(tr, text, occur, M, L, charSetSize);
}
