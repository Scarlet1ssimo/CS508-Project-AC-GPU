
#include <hip/hip_runtime.h>
#include <cassert>
#include <queue>
using std::queue;

int TrieBuildCPU(unsigned char* const* patterns, int* tr, int* idx, const int M, const int N, const int charSetSize) {
  int trieNodeNumber = 1;
  for (int i = 0; i < N; i++) {
    int state = 0;
    for (int j = 0; j < M; j++) {
      int c = patterns[i][j];
      if (!tr[state * charSetSize + c])
        tr[state * charSetSize + c] = trieNodeNumber++;
      state = tr[state * charSetSize + c];
    }
    idx[i] = state;
  }
  return trieNodeNumber;
}
void TrieReorder(int* tr, int* idx, const int N, const int trieNodeNumber, const int charSetSize) {
  // Reorder the trie by BFS order
  int* newTr   = (int*) malloc(sizeof(int) * trieNodeNumber * charSetSize);
  int* old2New = (int*) malloc(sizeof(int) * trieNodeNumber);
  queue<int> q;
  q.push(0);
  int cnt = 0;
  while (!q.empty()) {
    int u = q.front();
    q.pop();
    old2New[u] = cnt++;
    for (int i = 0; i < charSetSize; i++)
      if (auto v = tr[u * charSetSize + i])
        q.push(v);
  }
  assert(cnt == trieNodeNumber);
  for (int i = 0; i < trieNodeNumber; i++)
    for (int j = 0; j < charSetSize; j++) {
      assert(old2New[tr[i * charSetSize + j]] < trieNodeNumber);
      assert(tr[i * charSetSize + j] < trieNodeNumber);
      assert(old2New[i] < trieNodeNumber);
      assert(i < trieNodeNumber);
      newTr[old2New[i] * charSetSize + j] = old2New[tr[i * charSetSize + j]];
    }
  for (int i = 0; i < N; i++)
    idx[i] = old2New[idx[i]];
  memcpy(tr, newTr, sizeof(int) * trieNodeNumber * charSetSize);
  free(newTr);
  free(old2New);
}
void ACBuildCPU(int* tr, int* fail, int* postOrder, const int charSetSize) {
  queue<int> q;
  for (int c = 0; c < charSetSize; c++) {
    int state = tr[0 * charSetSize + c];
    if (state) {
      fail[state] = 0;
      q.push(state);
    }
  }
  int postOrderCnt = 0;
  while (!q.empty()) {
    int u = q.front();
    q.pop();
    postOrder[postOrderCnt++] = u;
    for (int i = 0; i < charSetSize; i++) {
      auto& v = tr[u * charSetSize + i];
      if (v) {
        fail[v] = tr[fail[u] * charSetSize + i];
        q.push(v);
      } else
        v = tr[fail[u] * charSetSize + i];
    }
  }
  // postOrderCnt == stateCnt-1, because state 0 is not included
}
void ACCPU(const int* tr, const unsigned char* text, int* occur, const int L, const int charSetSize) {
  int state = 0;
  for (int i = 0; i < L; i++) {
    state = tr[state * charSetSize + text[i]];
    occur[state]++;
  }
}
void ACPostCPU(int* out, const int* fail, const int* postOrder, const int postOrderCnt) {
  for (int i = postOrderCnt - 1; i >= 0; i--)
    out[fail[postOrder[i]]] += out[postOrder[i]];
}
